#include "hip/hip_runtime.h"
#include <iostream>
#include<cstdio>
#include<fstream>
#include<sstream>
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
#include<vector>
#include<cmath>
using namespace std;
const int N=29;
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
// Macro for timing kernel runs
#define START_METER {\
    hipEvent_t start, stop;\
    float elapsedTime;\
    hipEventCreate(&start);\
    hipEventRecord(start, 0);
#define STOP_METER hipEventCreate(&stop);\
    hipEventRecord(stop, 0);\
    hipEventSynchronize(stop);\
    hipEventElapsedTime(&elapsedTime, start, stop);\
    printf("Elapsed time : %f ms\n", elapsedTime);\
                }
bool sortinrev(const pair < double,vector<double> > &a,  
               const pair < double,vector<double> > &b) 
{ 
       return (a.first > b.first); 
}
__global__ 
void mean_col(double *data,double *mean )
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index<29)
    {
    for(int j = 0;j<24998;j++)
        {
        mean[index] = mean[index] + data[29*j+index];
        }
    mean[index]=double(mean[index]/24998.0);
    }
}

__global__ 
void std_col(double *data,double *mean,double *std){
        int index = threadIdx.x;
        double t = 0;
        for(int j = 0;j<24998;j++)
        {
            double t2 = data[29*j+index]- mean[index];
            t = t + t2*t2;
        }
        t= t/24998;
        std[index] = sqrt(t);
}
__global__
void data_normalize(double *data,double *mean,double *std){
    int index = threadIdx.x;
    for(int i = 0;i<24998;i++)
    {
            data[i*29+index] = (data[i*29+index]- mean[index])/std[index];
    }
}
__global__
void covariance(double *data,double *r){
    int index = threadIdx.x;
    for(int j=0;j<29;j++)
    {
        for(int k=0;k<24998;k++)
        {
            r[index*29+j] +=  (data[k*29+index])*data[k*29+j];
        }
    }
}
__global__ void gpu_matrix_transpose(double* mat_in, double* mat_out) 
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < 29 && idy < 24998) 
    {
        unsigned int pos = idy * 29 + idx;
        unsigned int trans_pos = idx * 24998 + idy;
        mat_out[trans_pos] = mat_in[pos];
    }
}



int main(void)
{
    time_t start, end; 
    start = clock();
    ios_base::sync_with_stdio(false);

    int col = 29;
    int rw = 24998;
    int row = 29;

    double *r1,*data,*d_data,*d_mean,*transpose,*d_t;

    r1 = (double*)calloc(row*N,sizeof(double));

    data = (double*)malloc(rw*col*sizeof(double));
    transpose = (double*)malloc(rw*col*sizeof(double));

    hipMalloc((void**)&d_data, rw*col*sizeof(double));
    hipMalloc((void**)&d_t, rw*col*sizeof(double));

    //********** file read ***********
    std::ifstream file("intrusion.csv");
    
    std::string lin;
    std::getline(file, lin);
    
    for(int rows = 0; rows < rw; ++rows)
    {

        string line;
        getline(file, line);
        if ( !file.good() )
            break;
        std::stringstream iss(line);
        for (int cols = 0; cols < col; ++cols)
        {
            std::string val;
            std::getline(iss, val, ',');
            if ( !iss.good() )
                break;

            std::stringstream convertor(val);
            convertor >> data[rows*col + cols];
        }
        ////cout<<endl;
    }

    //******** file read end *********




    double *mean;
    mean = (double*)calloc(col,sizeof(double));
    hipMalloc((void**)&d_mean, col*sizeof(double)); 
    hipMemcpy(d_data, data, rw*col*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_mean, mean, col*sizeof(double), hipMemcpyHostToDevice);
    mean_col <<<1, 29>>>(d_data,d_mean);
    //mean_col <<<1, 29>>>(d_mean);
    hipDeviceSynchronize();
    hipMemcpy(mean, d_mean, col*sizeof(double), hipMemcpyDeviceToHost);
    
    double *std,*d_std;
    std = (double*)malloc(col*sizeof(double));
    hipMalloc((void**)&d_std, col*sizeof(double)); 
    //hipMemcpy(data, d_data, rw*col*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(d_std, std, col*sizeof(double), hipMemcpyHostToDevice);
    std_col <<< 1,29 >>>(d_data,d_mean,d_std);
    hipDeviceSynchronize();
    hipMemcpy(data, d_data, rw*col*sizeof(double), hipMemcpyDeviceToHost);
    data_normalize <<< 1,29 >>>(d_data,d_mean,d_std);
    hipDeviceSynchronize();
    
    hipMemcpy(data, d_data, rw*col*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_std);
    hipFree(d_mean);
    free(mean);
    free(std);
    double *d_r;
    
    hipMalloc((void**)&d_r, N*col*sizeof(double));
    hipMemcpy(d_r, r1, N*col*sizeof(double), hipMemcpyHostToDevice);
    
    covariance<<< 1, 29 >>>(d_data,d_r);
    hipDeviceSynchronize();
    hipMemcpy(r1, d_r, N*col*sizeof(double), hipMemcpyDeviceToHost);
    ////cout<<"w"<<endl;

    
    
    double w[row];
    double v[row][N] ={0};
    double r[row][N]={0};
    for(int i=0;i<row;i++){
        for(int j=0;j<N;j++)
            r[i][j] = r1[i*N+j];
    }

    //cusolver *****************************
    hipsolverHandle_t handle;
    hipsolverDnCreate(&handle);
    int lwork;
    int rows = 29,cols=29;
    hipsolverDnDgesvd_bufferSize(
        handle,
        rows,
        cols,
        &lwork);

    double *d_A;
    hipMalloc(&d_A, sizeof(double)*N*cols);
    hipMemcpy(d_A, r1, sizeof(double)*N*cols, hipMemcpyHostToDevice);

    double *d_S;
    hipMalloc(&d_S, sizeof(double)*rows);

    double *d_U;
    hipMalloc(&d_U, sizeof(double)*rows*rows);

    double *d_VT,*VT,*S;

    VT = (double*)calloc(N*N,sizeof(double));
    S = (double*)calloc(N,sizeof(double));

    hipMalloc(&d_VT, sizeof(double)*rows*rows);

    double *d_work;
    hipMalloc(&d_work, sizeof(double)*lwork);

    double *d_rwork;
    hipMalloc(&d_rwork, sizeof(double)*(rows - 1));

    int *devInfo;
    hipMalloc(&devInfo, sizeof(int));

    for (int t = 0; t < 10; t++)
    {
        signed char jobu = 'A';
        signed char jobvt = 'A';
            hipsolverDnDgesvd(
            handle,
            jobu,
            jobvt,
            rows,
            cols,
            d_A,
            rows,
            d_S,
            d_U,
            rows,
            d_VT,
            rows,
            d_work,
            lwork,
            d_rwork,
            devInfo);
    }

    hipMemcpy(VT, d_VT, N*N*sizeof(double), hipMemcpyDeviceToHost);
    
    hipMemcpy(S, d_S, N*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_rwork);
    hipFree(d_S);
    hipFree(d_U);
    hipFree(d_VT);
    hipFree(d_work);

   /* for(int i=0;i<N;i++){
        std::cout<<S[i]<<"\n";
    }
*/
    //

    //******
    //sort(w.begin(),w.end());
    double eigen_sum  = 0;
    for(int i =0;i<row;i++)
    {    
        eigen_sum  += fabs(S[i]);
        //cout<<w[i]<<" ";   
    }
    //cout<<endl;

    double thres= 0.9;

    vector< vector<double> > v1(row, vector<double> (col,0));

    for(int i =0;i<row;i++)
    {
        for(int j=0;j<col;j++)
        {
            v1[i][j] = VT[i+29*j];
        }
    }

    vector< pair < double,vector<double> > > vc;

    for(int i =0;i<row;i++)
    {
        vc.push_back( make_pair(S[i],v1[i]) );

    }
    //sort(vc.begin(), vc.end(),sortinrev);

    double sum = 0;
    int dim = 0;
    for(int i =0;i<row;i++)
    {
        sum += fabs(vc[i].first);
        dim++;
        double temp = sum/eigen_sum;
        if(temp>0.9)
        {
            break;
        }

    }


    vector< vector<double> > red_data(rw, vector<double> (dim,0));


    for(int i =0;i<rw;i++)
    {
        for(int j=0;j<dim;j++)
        {
            for(int k=0;k<col;k++)
            {
                red_data[i][j] = red_data[i][j] + data[i*29+k]*(vc[j].second[k]);
            }
        }
    }

    end = clock();  
  
    double time_taken = double(end - start) / double(CLOCKS_PER_SEC); 
    cout << "Time taken by program is : " << fixed  
         << time_taken << setprecision(5); 
    cout << " sec " << endl; 

    ofstream outfile;
    outfile.open ("reduced_data.csv");
    for (int i = 0;i<dim-1;i++)
    {
        outfile<<"col"<<i<<",";
    }
    outfile<<"col"<<dim-1<<endl;

    for(int i =0;i<rw;i++)
    {
        for(int j=0;j<dim-1;j++)
        {
            outfile<<red_data[i][j]<<",";
        }
        outfile<<red_data[i][dim-1]<<endl;
    }

    outfile.close();
    return 0;
}



